#include "hip/hip_runtime.h"
#include <iostream>

#include "ray.h"
#include "vec3.h"
#include "hittable.h"
#include "sphere.h"
#include "hittable_list.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
//#include "const.h"
//#include "camera.h"
//#include "material.h"
//#include "texture.h"


__device__ color ray_color(const ray &r, hittable **world, int depth, hiprandState *st) {

    ray tmp_ray =r ;
    float color_scalar = 1.0;

    do{
        hit_record rec;
        if((*world)->hit(tmp_ray, 0.001, infinity, rec)){
            point3 target = rec.p + rec.normal + random_in_unit_sphere(st); 
            color_scalar = color_scalar * 0.5;
            tmp_ray = ray(rec.p, target - rec.p);
        }else {
            vec3 unit_direction = unit_vector(tmp_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            return  color_scalar*((1.0 - t)*color(1.0f, 1.0f, 1.0f) + t*color(0.5f, 0.7f, 1.0f));
        }
        depth--;
    } while(depth > 0);
    return color(0, 0, 0);

}

// list is a pointer to a list of all element
__global__ void create_world(hittable **list, hittable **world) {
    if(threadIdx.x == 0 && blockIdx.x == 0){
        *list = new sphere(vec3(0, 0, -1), 0.5);
        *(list + 1) = new sphere(vec3(0, -100.5, -1), 100);
        *world = new hittable_list(list, 2);
    }
}

__global__ void setup_random(int width, int height, hiprandState *st){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= width) || (j >= height)) return;
    int loc = j*width + i;
    hiprand_init(0xDEADBEEF, loc, 0, st + loc);
}

__global__ void render(color *buff, int width, int height, vec3 lower_left_corner, 
                        vec3 horizontal, vec3 vertical, vec3 origin, hittable **world, int spp, hiprandState *st){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= width) || (j >= height)) 
        return;
    int loc = j*width + i;
    color pixel(0, 0, 0);
    for(int s = 0; s < spp; s++) {
        float u = (float(i) + hiprand_uniform(st + loc)) / width;
        float v = (float(j) + hiprand_uniform(st + loc)) / height;
        ray r(origin, lower_left_corner + u*horizontal + v*vertical);
        pixel = pixel + ray_color(r, world, 50, st);
    }


    //buff[loc] = color(float(i)/width, float(j)/height, 0.2);
    buff[loc] = pixel / spp;
}

int main() {

    // Image :)
    const double aspect_ratio = 16.0/9.0;
    int image_width = 600;
    int image_height = image_width/aspect_ratio;
    int size = image_width * image_height;
    dim3 blocks(image_width/32+1,image_height/32+1);
    dim3 threads(32,32);
    
    hiprandState *dev_rand_states;
    hipMalloc((void**) &dev_rand_states, image_width*image_height*sizeof(hiprandState));


    color *cuda_buff;
    float viewport_height = 2.0;
    float viewport_width = viewport_height*aspect_ratio; 
    hittable **hit_list;
    hipMalloc((void**) &hit_list, 2*sizeof(hittable *));
    hittable **world;
    hipMalloc((void**) &world, sizeof(hittable *));



    setup_random<<<blocks, threads>>>(image_width, image_height, dev_rand_states);
    create_world<<<blocks, threads>>>(hit_list, world);
    
    hipError_t result = hipMallocManaged((void **) &cuda_buff, size*sizeof(color));
    if(result) {
        std::cerr << "Error allocating GPU memory: " << hipGetErrorString(result) << std::endl;
        exit(1);
    }
    render<<<blocks, threads>>>(cuda_buff, image_width, image_height, 
            vec3(-viewport_width/2, -viewport_height/2, -1), 
            vec3(4, 0, 0), vec3(0, 2, 0), vec3(0, 0, 0), world, 150, dev_rand_states);

    hipDeviceSynchronize();

    // Write buffer to ppm format and stdout
    std::cout << "P3\n" << image_width << ' ' << image_height<< "\n255" << std::endl;
    for(int j = image_height- 1; j >= 0; j--) {
        for(int i = 0; i < image_width; ++i) {
            int loc = j * image_width + i;
            float r = cuda_buff[loc].x();
            float g = cuda_buff[loc].y();
            float b = cuda_buff[loc].z();
            std::cout << int(255.99 *r) << " " << int(255.99 *g) << " " << int(255.99 *b) << std::endl;
        }
    }

    std::cerr << "\nDone!\n";
}
